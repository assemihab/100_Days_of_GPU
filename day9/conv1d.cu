#include <iostream>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 256  // Define block size for CUDA threads

__global__ void conv1d(const float *input, const float *kernel, float *output, int input_size, int kernel_size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < input_size - kernel_size + 1) {
        float sum = 0.0f;
        for (int j = 0; j < kernel_size; j++) {
            sum += input[idx + j] * kernel[j];
        }
        output[idx] = sum;
    }
}

void checkCudaError(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << msg << " - " << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }
}

int main() {
    const int input_size = 10;
    const int kernel_size = 3;
    const int output_size = input_size - kernel_size + 1;

    float h_input[input_size] = {1, 2, 3, 4, 5, 6, 7, 8, 9, 10};
    float h_kernel[kernel_size] = {1, 0, -1};
    float h_output[output_size];

    float *d_input, *d_kernel, *d_output;
    checkCudaError(hipMalloc(&d_input, input_size * sizeof(float)), "Allocating input");
    checkCudaError(hipMalloc(&d_kernel, kernel_size * sizeof(float)), "Allocating kernel");
    checkCudaError(hipMalloc(&d_output, output_size * sizeof(float)), "Allocating output");

    checkCudaError(hipMemcpy(d_input, h_input, input_size * sizeof(float), hipMemcpyHostToDevice), "Copying input");
    checkCudaError(hipMemcpy(d_kernel, h_kernel, kernel_size * sizeof(float), hipMemcpyHostToDevice), "Copying kernel");

    int grid_size = (output_size + BLOCK_SIZE - 1) / BLOCK_SIZE;
    conv1d<<<grid_size, BLOCK_SIZE>>>(d_input, d_kernel, d_output, input_size, kernel_size);
    checkCudaError(hipGetLastError(), "Kernel launch");
    checkCudaError(hipDeviceSynchronize(), "Kernel execution");

    checkCudaError(hipMemcpy(h_output, d_output, output_size * sizeof(float), hipMemcpyDeviceToHost), "Copying output");

    std::cout << "Output: ";
    for (int i = 0; i < output_size; i++) {
        std::cout << h_output[i] << " ";
    }
    std::cout << std::endl;

    hipFree(d_input);
    hipFree(d_kernel);
    hipFree(d_output);

    return 0;
}
