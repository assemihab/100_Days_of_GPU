
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>

using namespace std;

__global__ void encrypt(char *input, char *output, int key) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    output[idx] = input[idx] + key;
}

__global__ void decrypt(char *input, char *output, int key) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    output[idx] = input[idx] - key;
}


int main() 
{
    ifstream file("unencrypted.txt");
    string str;
    getline(file, str);
    file.close();
    char *din, *dout;
    char *input = new char[str.length()];
    char *output = new char[str.length()];
    strcpy(input, str.c_str());
    hipMalloc(&din, str.length());
    hipMalloc(&dout, str.length());

    hipMemcpy(din, input, str.length(), hipMemcpyHostToDevice);
    int blocksize=10;
    dim3 dimGrid(ceil(str.length()+blocksize-1/blocksize), 1, 1);
    dim3 dimBlock(blocksize, 1, 1);

    encrypt<<<dimGrid, dimBlock>>>(din, dout, 3);
    hipMemcpy(output, dout, str.length(), hipMemcpyDeviceToHost);
    cout << "Encrypted: " << output << endl;
    
    char*decrypted;
    hipMalloc(&decrypted, str.length());
    decrypt<<<dimGrid, dimBlock>>>(dout, decrypted, 3);
    hipMemcpy(output, decrypted, str.length(), hipMemcpyDeviceToHost);
    cout << "Decrypted: " << output << endl;
    return 0;

    }